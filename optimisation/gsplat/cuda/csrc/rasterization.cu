#include "hip/hip_runtime.h"
#include "bindings.h"
#include <hip/hip_cooperative_groups.h>
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>

namespace cg = cooperative_groups;

/****************************************************************************
 * Gaussian Tile Intersection
 ****************************************************************************/

__global__ void isect_tiles(
    // parallelize over N
    const uint32_t N,
    // data
    const float2 *__restrict__ means2d,              // [N, 2]
    const int32_t *__restrict__ radii,               // [N]
    const int64_t *__restrict__ cum_tiles_per_gauss, // [N]
    const uint32_t tile_size, const uint32_t tile_width, const uint32_t tile_height,
    int32_t *__restrict__ tiles_per_gauss, // [N]
    int64_t *__restrict__ isect_ids,       // [n_isects]
    int32_t *__restrict__ flatten_ids      // [n_isects]
) {
    // parallelize over N.
    uint32_t idx = cg::this_grid().thread_rank();
    bool first_pass = cum_tiles_per_gauss == nullptr;
    if (idx >=  N)
        return;
    if (radii[idx] <= 0) {
        if (first_pass)
            tiles_per_gauss[idx] = 0;
        return;
    }

    float tile_radius = radii[idx] / static_cast<float>(tile_size);
    float tile_x = means2d[idx].x / tile_size;
    float tile_y = means2d[idx].y / tile_size;

    // tile_min is inclusive, tile_max is exclusive
    uint2 tile_min, tile_max;
    tile_min.x = min(max(0, (uint32_t)floor(tile_x - tile_radius)), tile_width);
    tile_min.y = min(max(0, (uint32_t)floor(tile_y - tile_radius)), tile_height);
    tile_max.x = min(max(0, (uint32_t)ceil(tile_x + tile_radius)), tile_width);
    tile_max.y = min(max(0, (uint32_t)ceil(tile_y + tile_radius)), tile_height);

    if (first_pass) {
        // first pass only writes out tiles_per_gauss
        tiles_per_gauss[idx] =
            static_cast<int32_t>((tile_max.y - tile_min.y) * (tile_max.x - tile_min.x));
        return;
    }

    int64_t cur_idx = (idx == 0) ? 0 : cum_tiles_per_gauss[idx - 1];
    for (int32_t i = tile_min.y; i < tile_max.y; ++i) {
        for (int32_t j = tile_min.x; j < tile_max.x; ++j) {
            int64_t tile_id = i * tile_width + j;
            isect_ids[cur_idx] = tile_id;
            // the flatten index in [N]
            flatten_ids[cur_idx] = static_cast<int32_t>(idx);
            ++cur_idx;
        }
    }
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor>
isect_tiles_tensor(const torch::Tensor &means2d, // [N, 2]
                   const torch::Tensor &radii,   // [N]
                   const uint32_t tile_size,
                   const uint32_t tile_width, const uint32_t tile_height,
                   const bool sort, const bool double_buffer) {
    DEVICE_GUARD(means2d);
    CHECK_INPUT(means2d);
    CHECK_INPUT(radii);

    uint32_t N, nnz, total_elems;

    N = means2d.size(0); // number of gaussians
    total_elems = N;

    uint32_t n_tiles = tile_width * tile_height;
    at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();

    // first pass: compute number of tiles per gaussian
    torch::Tensor tiles_per_gauss =
        torch::empty_like(radii, radii.options().dtype(torch::kInt32));

    int64_t n_isects;
    torch::Tensor cum_tiles_per_gauss;
    if (total_elems) {
        isect_tiles<<<(total_elems + N_THREADS - 1) / N_THREADS, N_THREADS, 0,
                      stream>>>(
            N,
            (float2 *)means2d.data_ptr<float>(), radii.data_ptr<int32_t>(),
            nullptr, tile_size, tile_width, tile_height,
            tiles_per_gauss.data_ptr<int32_t>(), nullptr, nullptr);
        cum_tiles_per_gauss = torch::cumsum(tiles_per_gauss.view({-1}), 0);
        n_isects = cum_tiles_per_gauss[-1].item<int64_t>();
    } else {
        n_isects = 0;
    }

    // second pass: compute isect_ids and flatten_ids as a packed tensor
    torch::Tensor isect_ids =
        torch::empty({n_isects}, radii.options().dtype(torch::kInt64));
    torch::Tensor flatten_ids =
        torch::empty({n_isects}, radii.options().dtype(torch::kInt32));
    if (n_isects) {
        isect_tiles<<<(total_elems + N_THREADS - 1) / N_THREADS, N_THREADS, 0,
                      stream>>>(
            N,
            (float2 *)means2d.data_ptr<float>(), radii.data_ptr<int32_t>(),
            cum_tiles_per_gauss.data_ptr<int64_t>(),
            tile_size, tile_width, tile_height, nullptr,
            isect_ids.data_ptr<int64_t>(), flatten_ids.data_ptr<int32_t>());
    }

    // optionally sort the Gaussians by isect_ids
    if (n_isects && sort) {
        torch::Tensor isect_ids_sorted = torch::empty_like(isect_ids);
        torch::Tensor flatten_ids_sorted = torch::empty_like(flatten_ids);

        // https://nvidia.github.io/cccl/cub/api/structcub_1_1DeviceRadixSort.html
        // DoubleBuffer reduce the auxiliary memory usage from O(N+P) to O(P)
        if (double_buffer) {
            // Create a set of DoubleBuffers to wrap pairs of device pointers
            hipcub::DoubleBuffer<int64_t> d_keys(isect_ids.data_ptr<int64_t>(),
                                              isect_ids_sorted.data_ptr<int64_t>());
            hipcub::DoubleBuffer<int32_t> d_values(flatten_ids.data_ptr<int32_t>(),
                                                flatten_ids_sorted.data_ptr<int32_t>());
            CUB_WRAPPER(hipcub::DeviceRadixSort::SortPairs, d_keys, d_values, n_isects, 0, 64,
                        stream);
            switch (d_keys.selector) {
            case 0: // sorted items are stored in isect_ids
                isect_ids_sorted = isect_ids;
                break;
            case 1: // sorted items are stored in isect_ids_sorted
                break;
            }
            switch (d_values.selector) {
            case 0: // sorted items are stored in flatten_ids
                flatten_ids_sorted = flatten_ids;
                break;
            case 1: // sorted items are stored in flatten_ids_sorted
                break;
            }
            // printf("DoubleBuffer d_keys selector: %d\n", d_keys.selector);
            // printf("DoubleBuffer d_values selector: %d\n", d_values.selector);
        } else {
            CUB_WRAPPER(hipcub::DeviceRadixSort::SortPairs, isect_ids.data_ptr<int64_t>(),
                        isect_ids_sorted.data_ptr<int64_t>(),
                        flatten_ids.data_ptr<int32_t>(),
                        flatten_ids_sorted.data_ptr<int32_t>(), n_isects, 0, 64, stream);
        }
        return std::make_tuple(tiles_per_gauss, isect_ids_sorted, flatten_ids_sorted);
    } else {
        return std::make_tuple(tiles_per_gauss, isect_ids, flatten_ids);
    }
}

__global__ void isect_offset_encode(const uint32_t n_isects,
                                    const int64_t *__restrict__ isect_ids,
                                    const uint32_t n_tiles,
                                    int32_t *__restrict__ offsets // [n_tiles]
) {
    // e.g., ids: [1, 1, 1, 3, 3], n_tiles = 6
    // counts: [0, 3, 0, 2, 0, 0]
    // cumsum: [0, 3, 3, 5, 5, 5]
    // offsets: [0, 0, 3, 3, 5, 5]
    uint32_t idx = cg::this_grid().thread_rank();
    if (idx >= n_isects)
        return;

    int64_t id_curr = isect_ids[idx];

    if (idx == 0) {
        // write out the offsets until the first valid tile (inclusive)
        for (uint32_t i = 0; i < id_curr + 1; ++i)
            offsets[i] = static_cast<int32_t>(idx);
    }
    if (idx == n_isects - 1) {
        // write out the rest of the offsets
        for (uint32_t i = id_curr + 1; i < n_tiles; ++i)
            offsets[i] = static_cast<int32_t>(n_isects);
    }

    if (idx > 0) {
        // visit the current and previous isect_id and check if the (cid, tile_id)
        // pair changes.
        int64_t isect_id_prev = isect_ids[idx - 1];
        if (isect_id_prev == id_curr)
            return;

        // write out the offsets between the previous and current tiles
        int64_t id_prev = isect_id_prev;
        for (uint32_t i = id_prev + 1; i < id_curr + 1; ++i)
            offsets[i] = static_cast<int32_t>(idx);
    }
}

torch::Tensor isect_offset_encode_tensor(const torch::Tensor &isect_ids, // [n_isects]
                                         const uint32_t tile_width,
                                         const uint32_t tile_height) {
    DEVICE_GUARD(isect_ids);
    CHECK_INPUT(isect_ids);

    uint32_t n_isects = isect_ids.size(0);
    torch::Tensor offsets = torch::empty({tile_height, tile_width},
                                         isect_ids.options().dtype(torch::kInt32));
    if (n_isects) {
        uint32_t n_tiles = tile_width * tile_height;
        at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
        isect_offset_encode<<<(n_isects + N_THREADS - 1) / N_THREADS, N_THREADS, 0,
                              stream>>>(n_isects, isect_ids.data_ptr<int64_t>(),
                                        n_tiles, offsets.data_ptr<int32_t>());
    } else {
        offsets.fill_(0);
    }
    return offsets;
}

/****************************************************************************
 * Rasterization
 ****************************************************************************/

__global__ void rasterize_to_pixels_fwd_kernel(
    const uint32_t N, const uint32_t n_isects,
    const float2 *__restrict__ means2d,    // [N, 2]
    const float3 *__restrict__ conics,     // [N, 3]
    const float *__restrict__ amplitude,   // [N]
    const float *__restrict__ beta,        // [N]
    const uint32_t image_width, const uint32_t image_height, const uint32_t tile_size,
    const uint32_t tile_width, const uint32_t tile_height,
    const int32_t *__restrict__ tile_offsets, // [tile_height, tile_width]
    const int32_t *__restrict__ flatten_ids,  // [n_isects]
    float *__restrict__ render_colors // [image_height, image_width]
) {
    // each thread draws one pixel, but also timeshares caching gaussians in a
    // shared tile

    auto block = cg::this_thread_block();
    int32_t tile_id = block.group_index().y * tile_width + block.group_index().z;
    uint32_t i = block.group_index().y * tile_size + block.thread_index().y;
    uint32_t j = block.group_index().z * tile_size + block.thread_index().x;

    float px = (float)j + 0.5f;
    float py = (float)i + 0.5f;
    int32_t pix_id = i * image_width + j;

    // return if out of bounds
    // keep not rasterizing threads around for reading data
    bool inside = (i < image_height && j < image_width);
    bool done = !inside;

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    int32_t range_start = tile_offsets[tile_id];
    int32_t range_end =
        (tile_id == tile_width * tile_height - 1)
            ? n_isects
            : tile_offsets[tile_id + 1];
    const uint32_t block_size = block.size();
    uint32_t num_batches = (range_end - range_start + block_size - 1) / block_size;

    extern __shared__ int s[];
    int32_t *id_batch = (int32_t *)s;                              // [block_size]
    float2 *xy_batch = (float2 *)&id_batch[block_size];    // [block_size]
    float3 *conic_batch = (float3 *)&xy_batch[block_size]; // [block_size]
    float *amplitude_batch = (float *)&conic_batch[block_size]; // [block_size]
    float *beta_batch = (float *)&amplitude_batch[block_size]; // [block_size]

    // index of most recent gaussian to write to this thread's pixel
    uint32_t cur_idx = 0;

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing its
    // designated pixel
    uint32_t tr = block.thread_rank();

    float pix_out = 0.f;
    for (uint32_t b = 0; b < num_batches; ++b) {
        // resync all threads before beginning next batch
        // end early if entire tile is done
        if (__syncthreads_count(done) >= block_size) {
            break;
        }

        // each thread fetch 1 gaussian from front to back
        // index of gaussian to load
        uint32_t batch_start = range_start + block_size * b;
        uint32_t idx = batch_start + tr;
        if (idx < range_end) {
            int32_t g = flatten_ids[idx]; // flatten index in [N]
            id_batch[tr] = g;
            xy_batch[tr] = means2d[g];
            conic_batch[tr] = conics[g];
            amplitude_batch[tr] = amplitude[g];
            beta_batch[tr] = beta[g];
        }

        // wait for other threads to collect the gaussians in batch
        block.sync();

        // process gaussians in the current batch for this pixel
        uint32_t batch_size = min(block_size, range_end - batch_start);
        for (uint32_t t = 0; (t < batch_size) && !done; ++t) {
            const float3 conic = conic_batch[t];
            const float2 xy = xy_batch[t];
            const float2 delta = {xy.x - px, xy.y - py};
            const float sigma =
                0.5f * (conic.x * delta.x * delta.x + conic.z * delta.y * delta.y) +
                conic.y * delta.x * delta.y;
            float alpha = __expf(-__powf(sigma, beta_batch[t])) * amplitude_batch[t];
            if (sigma < 0.f) {
                continue;
            }

            int32_t g = id_batch[t];
            const float vis = alpha;
            pix_out += vis;

            cur_idx = batch_start + t;
        }
    }

    if (inside) {
        render_colors[pix_id] = pix_out;
    }
}

torch::Tensor rasterize_to_pixels_fwd_tensor(
    // Gaussian parameters
    const torch::Tensor &means2d,   // [N, 2]
    const torch::Tensor &conics,    // [N, 3]
    const torch::Tensor &amplitude, // [N]
    const torch::Tensor &beta,      // [N]
    // image size
    const uint32_t image_width, const uint32_t image_height, const uint32_t tile_size,
    // intersections
    const torch::Tensor &tile_offsets, // [tile_height, tile_width]
    const torch::Tensor &flatten_ids   // [n_isects]
) {
    DEVICE_GUARD(means2d);
    CHECK_INPUT(means2d);
    CHECK_INPUT(conics);
    CHECK_INPUT(amplitude);
    CHECK_INPUT(beta);
    CHECK_INPUT(tile_offsets);
    CHECK_INPUT(flatten_ids);

    uint32_t N = means2d.size(0); // number of gaussians
    uint32_t tile_height = tile_offsets.size(0);
    uint32_t tile_width = tile_offsets.size(1);
    uint32_t n_isects = flatten_ids.size(0);

    // Each block covers a tile on the image. In total there are
    // 1 * tile_height * tile_width blocks.
    dim3 threads = {tile_size, tile_size, 1};
    dim3 blocks = {1, tile_height, tile_width};

    torch::Tensor renders = torch::empty({image_height, image_width},
                                         means2d.options().dtype(torch::kFloat32));

    at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
    const uint32_t shared_mem =
        tile_size * tile_size * (sizeof(int32_t) + sizeof(float2) + sizeof(float3) + sizeof(float) + sizeof(float));

    if (hipFuncSetAttribute(reinterpret_cast<const void*>(rasterize_to_pixels_fwd_kernel),
                             hipFuncAttributeMaxDynamicSharedMemorySize,
                             shared_mem) != hipSuccess) {
        AT_ERROR("Failed to set maximum shared memory size (requested ", shared_mem,
                 " bytes), try lowering tile_size.");
    }
    rasterize_to_pixels_fwd_kernel<<<blocks, threads, shared_mem, stream>>>(
        N, n_isects, (float2 *)means2d.data_ptr<float>(),
        (float3 *)conics.data_ptr<float>(), (float *)amplitude.data_ptr<float>(),
        (float *)beta.data_ptr<float>(),
        image_width, image_height, tile_size, tile_width, tile_height,
        tile_offsets.data_ptr<int32_t>(), flatten_ids.data_ptr<int32_t>(),
        renders.data_ptr<float>());

    return renders;
}

__global__ void rasterize_to_pixels_bwd_kernel(
    const uint32_t N, const uint32_t n_isects,
    const float2 *__restrict__ means2d,    // [N, 2]
    const float3 *__restrict__ conics,     // [N, 3]
    const float *__restrict__ amplitude,   // [N]
    const float *__restrict__ beta,   // [N]
    const uint32_t image_width, const uint32_t image_height, const uint32_t tile_size,
    const uint32_t tile_width, const uint32_t tile_height,
    const int32_t *__restrict__ tile_offsets, // [tile_height, tile_width]
    const int32_t *__restrict__ flatten_ids,  // [n_isects]
    const float *__restrict__ render_colors_grad, // [image_height, image_width]
    float2 *__restrict__ means2d_grad,    // [N, 2]
    float3 *__restrict__ conics_grad,     // [N, 3]
    float *__restrict__ amplitude_grad,    // [N]
    float *__restrict__ beta_grad    // [N]
) {
    // each thread processes one pixel, but also timeshares caching gaussians in a
    // shared tile

    auto block = cg::this_thread_block();
    int32_t tile_id = block.group_index().y * tile_width + block.group_index().z;
    uint32_t i = block.group_index().y * tile_size + block.thread_index().y;
    uint32_t j = block.group_index().z * tile_size + block.thread_index().x;

    float px = (float)j + 0.5f;
    float py = (float)i + 0.5f;
    int32_t pix_id = i * image_width + j;

    // return if out of bounds
    bool inside = (i < image_height && j < image_width);
    bool done = !inside;

    // have all threads in tile process the same gaussians in batches
    int32_t range_start = tile_offsets[tile_id];
    int32_t range_end =
        (tile_id == tile_width * tile_height - 1)
            ? n_isects
            : tile_offsets[tile_id + 1];
    const uint32_t block_size = block.size();
    uint32_t num_batches = (range_end - range_start + block_size - 1) / block_size;

    extern __shared__ int s[];
    int32_t *id_batch = (int32_t *)s;                              // [block_size]
    float2 *xy_batch = (float2 *)&id_batch[block_size];    // [block_size]
    float3 *conic_batch = (float3 *)&xy_batch[block_size]; // [block_size]
    float *amplitude_batch = (float *)&conic_batch[block_size]; // [block_size]
    float *beta_batch = (float *)&amplitude_batch[block_size]; // [block_size]

    // index of most recent gaussian to write to this thread's pixel
    uint32_t cur_idx = 0;

    // collect and process batches of gaussians
    uint32_t tr = block.thread_rank();

    float grad_out = inside ? render_colors_grad[pix_id] : 0.f;
    for (uint32_t b = 0; b < num_batches; ++b) {
        if (__syncthreads_count(done) >= block_size) {
            break;
        }

        uint32_t batch_start = range_start + block_size * b;
        uint32_t idx = batch_start + tr;
        if (idx < range_end) {
            int32_t g = flatten_ids[idx];
            id_batch[tr] = g;
            xy_batch[tr] = means2d[g];
            conic_batch[tr] = conics[g];
            amplitude_batch[tr] = amplitude[g];
            beta_batch[tr] = beta[g];
        }

        block.sync();

        uint32_t batch_size = min(block_size, range_end - batch_start);
        for (uint32_t t = 0; (t < batch_size) && !done; ++t) {
            const float3 conic = conic_batch[t];
            const float2 xy = xy_batch[t];
            const float2 delta = {xy.x - px, xy.y - py};
            const float sigma =
                0.5f * (conic.x * delta.x * delta.x + conic.z * delta.y * delta.y) +
                conic.y * delta.x * delta.y;
            float pow = __powf(sigma, beta_batch[t]);
            float exp = __expf(-pow);
            float I = exp * amplitude_batch[t];

            if (sigma < 0.f) {
                continue;
            }

            int32_t g = id_batch[t];
            float vis_grad = grad_out;

            float I_grad = vis_grad;
            float p_grad = I_grad * -exp * amplitude_batch[t];
            float sigma_grad = p_grad * beta_batch[t] * __powf(sigma, beta_batch[t]-1);
            float2 delta_grad;
            delta_grad.x = sigma_grad * (conic.x * delta.x + conic.y * delta.y);
            delta_grad.y = sigma_grad * (conic.z * delta.y + conic.y * delta.x);

            float2 xy_grad;
            xy_grad.x = delta_grad.x;
            xy_grad.y = delta_grad.y;

            atomicAdd(&means2d_grad[g].x, xy_grad.x);
            atomicAdd(&means2d_grad[g].y, xy_grad.y);
            atomicAdd(&conics_grad[g].x, sigma_grad * 0.5f * delta.x * delta.x);
            atomicAdd(&conics_grad[g].y, sigma_grad * delta.x * delta.y);
            atomicAdd(&conics_grad[g].z, sigma_grad * 0.5f * delta.y * delta.y);
            atomicAdd(&amplitude_grad[g], I_grad * exp);
            atomicAdd(&beta_grad[g], I_grad * p_grad * pow * __logf(sigma));

            cur_idx = batch_start + t;
        }
    }
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
rasterize_to_pixels_bwd_tensor(
    // Gaussian parameters
    const torch::Tensor &means2d,                   // [N, 2]
    const torch::Tensor &conics,                    // [N, 3]
    const torch::Tensor &amplitude,                 // [N]
    const torch::Tensor &beta,                 // [N]
    // image size
    const uint32_t image_width, const uint32_t image_height, const uint32_t tile_size,
    // intersections
    const torch::Tensor &tile_offsets, // [tile_height, tile_width]
    const torch::Tensor &flatten_ids,  // [n_isects]
    // gradients of outputs
    const torch::Tensor &grad_render // [image_height, image_width]
    ) {
    DEVICE_GUARD(means2d);
    CHECK_INPUT(means2d);
    CHECK_INPUT(conics);
    CHECK_INPUT(beta);
    CHECK_INPUT(tile_offsets);
    CHECK_INPUT(flatten_ids);
    CHECK_INPUT(grad_render);

    uint32_t N = means2d.size(0); // number of gaussians
    uint32_t n_isects = flatten_ids.size(0);
    uint32_t tile_height = tile_offsets.size(0);
    uint32_t tile_width = tile_offsets.size(1);

    // Each block covers a tile on the image. In total there are
    // tile_height * tile_width blocks.
    dim3 threads = {tile_size, tile_size, 1};
    dim3 blocks = {1, tile_height, tile_width};

    torch::Tensor grad_means2d = torch::zeros_like(means2d);
    torch::Tensor grad_conics = torch::zeros_like(conics);
    torch::Tensor grad_amplitude = torch::zeros_like(amplitude);
    torch::Tensor grad_beta = torch::zeros_like(beta);

    if (n_isects) {
        const uint32_t shared_mem = tile_size * tile_size *
                                    (sizeof(int32_t) + sizeof(float2) + sizeof(float3) +
                                     sizeof(float) + sizeof(float));
        at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
        if (hipFuncSetAttribute(reinterpret_cast<const void*>(rasterize_to_pixels_bwd_kernel),
                                 hipFuncAttributeMaxDynamicSharedMemorySize,
                                 shared_mem) != hipSuccess) {
            AT_ERROR("Failed to set maximum shared memory size (requested ",
                     shared_mem, " bytes), try lowering tile_size.");
        }
        rasterize_to_pixels_bwd_kernel<<<blocks, threads, shared_mem, stream>>>(
            N, n_isects, (float2 *)means2d.data_ptr<float>(),
            (float3 *)conics.data_ptr<float>(), amplitude.data_ptr<float>(),
            beta.data_ptr<float>(),
            image_width, image_height, tile_size, tile_width, tile_height,
            tile_offsets.data_ptr<int32_t>(), flatten_ids.data_ptr<int32_t>(),
            grad_render.data_ptr<float>(),
            (float2 *)grad_means2d.data_ptr<float>(),
            (float3 *)grad_conics.data_ptr<float>(),
            grad_amplitude.data_ptr<float>(),
            grad_beta.data_ptr<float>()
            );
    }

    return std::make_tuple(grad_means2d, grad_conics, grad_amplitude, grad_beta);
}
